#include "hip/hip_runtime.h"
%%cu

#include <iostream>
#include <sstream>
#include <string>
#include <fstream>
#include <bits/stdc++.h>
#include <math.h>

#define N 453
#define BLOCKSIZE 960

using namespace std;    

__global__ void calculosLuminosidad(float *lB, float *lM, float* lA, float* lR)
{
    int myID = blockIdx.x * blockDim.x + threadIdx.x;
    if(myID < N){
        lR[myID] = (lB[myID] + lM[myID] + lA[myID])/3;
    }
    
}



int main(int argc, char** argv) 
{

    int size = sizeof(float) * N;

    float* hst_luminosidadBaja = (float *)malloc(size);
    float* hst_bpmBaja = (float *)malloc(size);
    float* hst_luminosidadMedia = (float *)malloc(size);
    float* hst_bpmMedia = (float *)malloc(size);
    float* hst_luminosidadAlta = (float *)malloc(size);
    float* hst_bpmAlta = (float *)malloc(size);
    float* hst_res = (float *)malloc(size);
    float* hst_res2 = (float *)malloc(size);
    
    
    float *dev_luminosidadBaja,  *dev_luminosidadMedia, *dev_luminosidadAlta, *dev_res;
    hipMalloc((void **)&dev_luminosidadBaja, size);
    hipMalloc((void **)&dev_luminosidadMedia, size);
    hipMalloc((void **)&dev_luminosidadAlta, size);
    hipMalloc((void **)&dev_res, size);

    float *dev_bpmBaja, *dev_bpmMedia, *dev_bpmAlta, *dev_res2;
    hipMalloc((void **)&dev_bpmBaja, size);
    hipMalloc((void **)&dev_bpmMedia, size);
    hipMalloc((void **)&dev_bpmAlta, size);
    hipMalloc((void **)&dev_res2, size);

    string linea;        
    string luminosidadBaja, bpmBaja, luminosidadMedia, bpmMedia, luminosidadAlta, bpmAlta;    
                                                               
    ifstream archivo ("datosFinales.csv"); 

    if (archivo.fail()) {    
        cerr << "No es posible abrir el archivo" << endl;         
        return 1;
    }
    
    getline(archivo,linea);                            

    int i = 0;
    while (getline(archivo,linea)) { 

        stringstream stream(linea);

        getline(stream, luminosidadBaja, ';');
        getline(stream, bpmBaja, ';');
        getline(stream  , luminosidadMedia, ';');
        getline(stream  , bpmMedia, ';');
        getline(stream  , luminosidadAlta, ';');
        getline(stream  , bpmAlta, ';');

        try{
            if (luminosidadBaja != ""){
                hst_luminosidadBaja[i] = stof(luminosidadBaja);
                hst_bpmBaja[i] = stof(bpmBaja);
            }else{
                hst_luminosidadBaja[i] = 0;
                hst_bpmBaja[i] = 0;
            }
            if (luminosidadMedia != ""){
                hst_luminosidadMedia[i] = stof(luminosidadMedia);
                hst_bpmMedia[i] = stof(bpmMedia);
            }else{
                hst_luminosidadMedia[i] = 0;
                hst_bpmMedia[i] = 0;
            }
            if (luminosidadAlta != ""){
                hst_luminosidadAlta[i] = stof(luminosidadAlta);
                hst_bpmAlta[i] = stof(bpmAlta);
            } else{
                hst_luminosidadAlta[i] = 0;
                hst_bpmAlta[i] = 0;
            }
            
        }catch(const std::invalid_argument& ia){
            cout << "Argumento Invalido: " << ia.what() << endl;
        }
        
        i++;
    }


    archivo.close();    
    

    hipMemcpy(dev_luminosidadBaja, hst_luminosidadBaja, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_luminosidadMedia, hst_luminosidadMedia, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_luminosidadAlta, hst_luminosidadAlta, size, hipMemcpyHostToDevice);

    hipMemcpy(dev_bpmBaja, hst_bpmBaja, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_bpmMedia, hst_bpmMedia, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_bpmAlta, hst_bpmAlta, size, hipMemcpyHostToDevice);


    int threadsPerBlock = BLOCKSIZE;
    int temp = N + threadsPerBlock - 1;
    int blocksPerGrid = temp / threadsPerBlock;

    calculosLuminosidad<<<blocksPerGrid, threadsPerBlock>>>(dev_luminosidadBaja, dev_luminosidadMedia, dev_luminosidadAlta, dev_res);
    calculosLuminosidad<<<blocksPerGrid, threadsPerBlock>>>(dev_bpmBaja, dev_bpmMedia, dev_bpmAlta, dev_res2);

    hipMemcpy(hst_res, dev_res, size, hipMemcpyDeviceToHost);
    hipMemcpy(hst_res2, dev_res2, size, hipMemcpyDeviceToHost);


    float promedio1 = 0.0;
    for(int i = 0; i < N; i++) {
        promedio1 += hst_res[i];
    }
    promedio1 = promedio1 /  N;

    cout << "El promedio de la luminosidad es: " << promedio1 << endl;

    if(promedio1 <= 450){
        cout << "La luminosidad es alta" << endl;
    }else if(promedio1 > 450 && promedio1 <= 620){
        cout << "La luminosidad es media" << endl;
    }else if(promedio1 > 620){
        cout << "La luminosidad es baja" << endl;
    }



    
    float promedio2 = 0.0;
    for(int i = 0; i < N; i++) {
        promedio2 += hst_res2[i];
    }
    promedio2 = promedio2 /  N;

    cout << "El promedio del bpm es: " << promedio2 << endl;

    if(promedio2 <= 60){
        cout << "El bpm es bajo" << endl;
    }else if(promedio2 > 60 && promedio2 <= 100){
        cout << "El bpm es medio" << endl;
    }else if(promedio2 > 100){
        cout << "El bpm es alto" << endl;
    }

    cout << "Si sus bpm estan fuera de lo normal, lo recomendado es visitar un medico :)" << endl;


    //libera memoria del host
    free(hst_luminosidadBaja);
    free(hst_bpmBaja);
    free(hst_luminosidadMedia);
    free(hst_bpmMedia);
    free(hst_luminosidadAlta);
    free(hst_bpmAlta);
    free(hst_res);

    hipFree(dev_luminosidadBaja);
    hipFree(dev_luminosidadMedia);
    hipFree(dev_luminosidadAlta);
    hipFree(dev_res);

    hipFree(dev_bpmBaja);
    hipFree(dev_bpmMedia);
    hipFree(dev_bpmAlta);
    hipFree(dev_res2);

    return 0; 
}